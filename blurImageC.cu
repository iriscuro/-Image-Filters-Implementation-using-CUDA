#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

#define BLUR_SIZE 5
#define BLOCK_SIZE 16

__global__ void blurKernel(unsigned char* in, unsigned char* out, int w, int h) {
    int Col = threadIdx.x + blockIdx.x * blockDim.x;
    int Row = threadIdx.y + blockIdx.y * blockDim.y;

    if (Col < w && Row < h) {
        int pixR = 0;
        int pixG = 0;
        int pixB = 0;
        int pixels = 0;

        for (int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) {
            for (int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) {
                int curRow = Row + blurRow;
                int curCol = Col + blurCol;

                if (curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
                    pixR += in[(curRow * w + curCol) * 3];
                    pixG += in[(curRow * w + curCol) * 3 + 1];
                    pixB += in[(curRow * w + curCol) * 3 + 2];
                    pixels++;
                }
            }
        }

        out[(Row * w + Col) * 3] = static_cast<unsigned char>(pixR / pixels);
        out[(Row * w + Col) * 3 + 1] = static_cast<unsigned char>(pixG / pixels);
        out[(Row * w + Col) * 3 + 2] = static_cast<unsigned char>(pixB / pixels);
    }
}

int main() {
    cv::Mat image = cv::imread("input.jpg", cv::IMREAD_COLOR);
    int width = image.cols;
    int height = image.rows;

    unsigned char* h_in = image.data;
    unsigned char* h_out = new unsigned char[width * height * 3];
    unsigned char* d_in;
    unsigned char* d_out;

    hipMalloc((void**)&d_in, width * height * 3 * sizeof(unsigned char));
    hipMalloc((void**)&d_out, width * height * 3 * sizeof(unsigned char));

    hipMemcpy(d_in, h_in, width * height * 3 * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((width + BLOCK_SIZE - 1) / BLOCK_SIZE, (height + BLOCK_SIZE - 1) / BLOCK_SIZE);

    blurKernel<<<gridDim, blockDim>>>(d_in, d_out, width, height);

    hipMemcpy(h_out, d_out, width * height * 3 * sizeof(unsigned char), hipMemcpyDeviceToHost);

    cv::Mat output(height, width, CV_8UC3, h_out);
    cv::imwrite("output.jpg", output);

    hipFree(d_in);
    hipFree(d_out);
    delete[] h_out;

    return 0;
}